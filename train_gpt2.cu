#include "hip/hip_runtime.h"
/*
GPT-2 Transformer Neural Net training loop. See README.md for usage.
*/
#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string>
#include <string_view>
#include <sys/stat.h>
#include <sys/types.h>
#include <mpi.h> // MODIFICACION: Incluir cabecera de MPI
// ----------- CPU utilities -----------
// defines: fopenCheck, freadCheck, fcloseCheck, fseekCheck, mallocCheck
// defines: create_dir_if_not_exists, find_max_step, ends_with_bin
#include "llmc/utils.h"
// defines: tokenizer_init, tokenizer_decode, tokenizer_free
#include "llmc/tokenizer.h"
// defines: dataloader_init, dataloader_reset, dataloader_next_batch, dataloader_free
// defines: evalloader_init, evalloader_reset, evalloader_next_batch, evalloader_free
#include "llmc/dataloader.h"
// defines: manual_seed, normal_ (same as torch.manual_seed and torch.normal)
#include "llmc/rand.h"
// defines: lr_scheduler_init, get_learning_rate
#include "llmc/schedulers.h"
// defines: sample_softmax, random_f32
#include "llmc/sampler.h"
// defines: logger_init, logger_log_eval, logger_log_val, logger_log_train
#include "llmc/logger.h"
// defines: get_flops_promised
#include "llmc/mfu.h"
// defines: OutlierDetector, init_detector, update_detector
#include "llmc/outlier_detector.h"
// ----------- GPU utilities -----------
// defines:
// WARP_SIZE, MAX_1024_THREADS_BLOCKS, CEIL_DIV, cudaCheck, PRECISION_MODE
// NVTX_RANGE_FN
#include "llmc/cuda_common.h"
// defines:
// Packed128, f128, x128
// warpReduceSum, warpReduceMax, blockReduce, copy_and_cast_kernel, cudaMallocConditionallyManaged
#include "llmc/cuda_utils.cuh"
// defines: CUBLAS_LOWP, cublasCheck, cublaslt_workspace_size, cublaslt_workspace
// defines: cublas_compute, cublaslt_handle, cublas_handle
#include "llmc/cublas_common.h"
// ----------- Layer implementations in CUDA -----------
// defines: encoder_forward, encoder_backward
#include "llmc/encoder.cuh"
// defines: layernorm_forward, residual_forward, fused_residual_forward5, layernorm_backward
#include "llmc/layernorm.cuh"
// defines: matmul_cublaslt, matmul_forward, matmul_backward, gelu_forward, gelu_backward_inplace
#include "llmc/matmul.cuh"
#ifdef ENABLE_CUDNN
// defines: create_cudnn, destroy_cudnn, attention_forward_cudnn, attention_backward_cudnn
#include "llmc/cudnn_att.h"
#else
// defines: attention_forward, attention_backward
#include "llmc/attention.cuh"
#endif
// defines: fused_classifier
#include "llmc/fused_classifier.cuh"
// defines: adamw_kernel3
#include "llmc/adamw.cuh"
// defines: global_norm_squared
#include "llmc/global_norm.cuh"
// ----------- Multi-GPU support -----------
// defines: ncclFloatX, ncclCheck, MultiGpuConfig, ShardInfo
// defines: printf0, multi_gpu_config
// defines: multi_gpu_config_init, multi_gpu_config_free
// defines: set_zero_configs, multi_gpu_cpu_float_sum, multi_gpu_barrier
// defines: multi_gpu_get_shard_offset, multi_gpu_async_reduce_gradient
#include "llmc/zero.cuh"

// ----------------------------------------------------------------------------
// global vars for I/O
char filename_buffer[512];

// ----------------------------------------------------------------------------
// global vars containing information about the GPU this process is running on
hipDeviceProp_t deviceProp; // fills in common_start()
hipStream_t main_stream;
// buffer size to use for device <-> disk io
constexpr const size_t IO_BUF_SIZE = 32 * 1024 * 1024;

// ----------------------------------------------------------------------------
// GPT-2 model definition

typedef struct {
    int max_seq_len; // max sequence length, e.g. 1024
    int vocab_size; // vocab size, e.g. 50257
    int padded_vocab_size; // padded to e.g. %128==0, 50304
    int num_layers; // number of layers, e.g. 12
    int num_heads; // number of heads in attention, e.g. 12
    int channels; // number of channels, e.g. 768
} GPT2Config;

// the parameters of the model
constexpr const int NUM_PARAMETER_TENSORS = 16;
typedef struct {
    floatX* wte; // (V, C)
    floatX* wpe; // (maxT, C)
    floatX* ln1w; // (L, C)
    floatX* ln1b; // (L, C)
    floatX* qkvw; // (L, 3*C, C)
    floatX* qkvb; // (L, 3*C)
    floatX* attprojw; // (L, C, C)
    floatX* attprojb; // (L, C)
    floatX* ln2w; // (L, C)
    floatX* ln2b; // (L, C)
    floatX* fcw; // (L, 4*C, C)
    floatX* fcb; // (L, 4*C)
    floatX* fcprojw; // (L, C, 4*C)
    floatX* fcprojb; // (L, C)
    floatX* lnfw; // (C)
    floatX* lnfb; // (C)
} ParameterTensors;
static_assert(sizeof(ParameterTensors) == NUM_PARAMETER_TENSORS * sizeof(void*), "Inconsistent sizes!");

void fill_in_parameter_sizes(size_t* param_sizes, size_t* param_sizeof, GPT2Config config) {
    size_t Vp = config.padded_vocab_size;
    size_t C = config.channels;
    size_t maxT = config.max_seq_len;
    size_t L = config.num_layers;
    param_sizes[0] = Vp * C; // wte
    param_sizes[1] = maxT * C; // wpe
    param_sizes[2] = L * C; // ln1w
    param_sizes[3] = L * C; // ln1b
    param_sizes[4] = L * (3 * C) * C; // qkvw
    param_sizes[5] = L * (3 * C); // qkvb
    param_sizes[6] = L * C * C; // attprojw
    param_sizes[7] = L * C; // attprojb
    param_sizes[8] = L * C; // ln2w
    param_sizes[9] = L * C; // ln2b
    param_sizes[10] = L * (4 * C) * C; // fcw
    param_sizes[11] = L * (4 * C); // fcb
    param_sizes[12] = L * C * (4 * C); // fcprojw
    param_sizes[13] = L * C; // fcprojb
    param_sizes[14] = C; // lnfw
    param_sizes[15] = C; // lnfb

    // populate the parameter sizes in bytes (all the same for now, keeping for future use)
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        param_sizeof[i] = sizeof(floatX);
    }
}

// allocate memory for the parameters and point the individual tensors to the right places
void* malloc_and_point_parameters(ParameterTensors* params, size_t* param_elements, size_t *param_sizeof) {
    // calculate the total number of parameters and bytes across all tensors
    size_t num_parameters_bytes = 0;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        num_parameters_bytes += param_elements[i] * param_sizeof[i];
    }
    // malloc all parameters all at once on the device
    void* params_memory;
    cudaCheck(hipMalloc((void**)&params_memory, num_parameters_bytes));
    // assign all the tensors their place in the array
    floatX** ptrs[] = {
        &params->wte, &params->wpe, &params->ln1w, &params->ln1b, &params->qkvw, &params->qkvb,
        &params->attprojw, &params->attprojb, &params->ln2w, &params->ln2b, &params->fcw, &params->fcb,
        &params->fcprojw, &params->fcprojb, &params->lnfw, &params->lnfb
    };
    char* params_memory_iterator = (char*)params_memory;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        *(ptrs[i]) = (floatX*)params_memory_iterator;
        params_memory_iterator += param_elements[i] * param_sizeof[i];
    }
    return params_memory;
}

constexpr int NUM_ACTIVATION_TENSORS = 21;
typedef struct {
    floatX* encoded; // (B, T, C)
    floatX* ln1; // (L, B, T, C)
    float* ln1_mean; // (L, B, T)
    float* ln1_rstd; // (L, B, T)
    floatX* atty; // (L, B, T, C)
    // cuDNN saves only some statistics information
#if ENABLE_CUDNN
    float* att;  // (L, B, NH, T)
#else
    floatX* att; // (L, B, NH, T, T)
#endif

    floatX* residual2; // (L, B, T, C)
    floatX* ln2; // (L, B, T, C)
    float* ln2_mean; // (L, B, T)
    float* ln2_rstd; // (L, B, T)
    floatX* fch; // (L, B, T, 4*C)
    floatX* fch_gelu; // (L, B, T, 4*C)
    floatX* residual3; // (L, B, T, C)
    floatX* lnf; // (B, T, C);   if LN recomputation is enabled (-r 2 and above), will be used for _all_ layernorms
    float* lnf_mean; // (B, T)
    float* lnf_rstd; // (B, T)
    float* losses; // (B, T), will be accumulated in micro-steps
    // adding these two compared to the CPU .c code, needed for attention kernel as buffers
    floatX* qkvr; // (L, B, T, 3*C)
    // in inference mode, this buffer will store the logits
    // in training mode, this buffer will contain the *gradients* of the logits.
    // during the processing of transformer blocks, we will also use this as a
    // general scratchpad buffer. Allocation is made large enough to hold (B, T, 3C),
    // (B, NH, T, T), and (B, T, V) shaped tensors.
    floatX* output;

    // some additional scratch buffers
    floatX* scratch_bt4c;   // (B, T, 4*C)
    floatX* scratch_btc;    // (B, T, C)
} ActivationTensors;


struct TensorSpec {
    void** ptr;
    size_t size;
    DType type;
};


#define TENSOR_SPEC(pointer, size) TensorSpec{(void**)(&pointer), (size), dtype_of(pointer)};

void fill_in_activation_sizes(const ActivationTensors* data, TensorSpec (&tensors)[NUM_ACTIVATION_TENSORS], size_t B, size_t T, GPT2Config config, int recompute) {
    size_t Vp = config.padded_vocab_size;
    size_t L = config.num_layers;
    size_t NH = config.num_heads;
    size_t C = config.channels;
    tensors[0] = TENSOR_SPEC(data->encoded, B * T * C);
    // if recompute >= 1 then we will recompute the layernorm forward activation during backward pass
    tensors[1] = TENSOR_SPEC(data->ln1,  (recompute < 2) ? L * B * T * C : 0);
    tensors[2] = TENSOR_SPEC(data->ln1_mean, L * B * T);
    tensors[3] = TENSOR_SPEC(data->ln1_rstd, L * B * T);
    tensors[4] = TENSOR_SPEC(data->atty, L * B * T * C);
    #ifdef ENABLE_CUDNN
    // FP32 stats tensor for cuDNN to be passed to backward pass
    tensors[5] = TENSOR_SPEC(data->att, L * B * NH * T);
    #else
    tensors[5] = TENSOR_SPEC(data->att, L * B * NH * T * T);
    #endif
    tensors[6] = TENSOR_SPEC(data->residual2, L * B * T * C);
    // if recompute >= 1 then we will recompute the layernorm forward activation during backward pass
    tensors[7] = TENSOR_SPEC(data->ln2, (recompute < 2) ? L * B * T * C : 0);
    tensors[8] = TENSOR_SPEC(data->ln2_mean, L * B * T);
    tensors[9] = TENSOR_SPEC(data->ln2_rstd, L * B * T);
    tensors[10] = TENSOR_SPEC(data->fch, L * B * T * 4*C);
    // if recompute >= 1 then we will recompute gelu_forward during backward and use this as scratch buffer
    tensors[11] = TENSOR_SPEC(data->fch_gelu, (recompute < 1) ? L * B * T * 4*C : B * T * 4*C);
    tensors[12] = TENSOR_SPEC(data->residual3, L * B * T * C);
    tensors[13] = TENSOR_SPEC(data->lnf, B * T * C);
    tensors[14] = TENSOR_SPEC(data->lnf_mean, B * T);
    tensors[15] = TENSOR_SPEC(data->lnf_rstd, B * T);
    tensors[16] = TENSOR_SPEC(data->losses, B * T);
    tensors[17] = TENSOR_SPEC(data->qkvr, L * B * T * 3*C);
    tensors[18] = TENSOR_SPEC(data->output, B * T * max(3*C, max(NH*T, Vp)));

    tensors[19] = TENSOR_SPEC(data->scratch_bt4c, B * T * 4 * C);
    tensors[20] = TENSOR_SPEC(data->scratch_btc, B * T * C);
}

void* malloc_and_point_activations(TensorSpec (&tensors)[NUM_ACTIVATION_TENSORS]) {
    size_t bytes = 0;
    for (size_t i = 0; i < NUM_ACTIVATION_TENSORS; i++) {
        bytes += tensors[i].size * sizeof_dtype(tensors[i].type);
    }

    printf0("allocating %d MiB for activations\n", (int)round(bytes / (1024 * 1024)));

    void* acts_memory;
    cudaCheck(hipMalloc((void**)&acts_memory, bytes));

    // hipMalloc does not guarantee initial memory values so we memset the allocation here
    // this matters because e.g. non-cuDNN attention assumes the attention buffer is zeroed
    // todo - up to ~100ms on slow GPUs, could theoretically be more selective, but this is safer
    cudaCheck(hipMemset(acts_memory, 0, bytes));

    char* acts_memory_iterator = (char*)acts_memory;
    for (size_t i = 0; i < NUM_ACTIVATION_TENSORS; i++) {
        // extra protection so we don't accidentally use an empty buffer
        if(tensors[i].size == 0) {
            *(tensors[i].ptr) = NULL;
        }else {
            *(tensors[i].ptr) = acts_memory_iterator;
            acts_memory_iterator += tensors[i].size * sizeof_dtype(tensors[i].type);
        }
    }
    return acts_memory;
}

typedef struct {
    GPT2Config config;
    // the weights of the model, and their sizes
    ParameterTensors params;
    size_t param_elements[NUM_PARAMETER_TENSORS];
    size_t param_sizeof[NUM_PARAMETER_TENSORS];
    void* params_memory;
    size_t num_parameters;
    size_t num_parameters_bytes;
    // gradients of the weights
    ParameterTensors grads;
    void* grads_memory;
    // buffers for the AdamW optimizer
    float* m_memory;
    float* v_memory;
    float* master_weights;     // is NULL unless fp32 weights is enabled.
    // the activations of the model, and their sizes
    ActivationTensors acts;
    TensorSpec acts_specs[NUM_ACTIVATION_TENSORS];
    void* acts_memory;
    // other run state configuration
    int batch_size; // the batch size (B) of current forward pass
    int seq_len; // the sequence length (T) of current forward pass
    int* inputs; // the input tokens for the current forward pass
    int* targets; // the target tokens for the current forward pass
    float mean_loss; // after the last backward micro-batch, will be populated with mean loss across all GPUs and micro-steps
    float* accumulated_mean_loss; // GPU buffer used to accumulate loss across micro-steps
    float* cpu_losses; // CPU buffer to copy the losses to, allocated with hipHostMalloc
    unsigned long long rng_state; // the RNG state for seeding stochastic rounding etc.
    unsigned long long rng_state_last_update; // RNG before last gpt2_update() to re-round identically from master weights
    int use_master_weights; // keep master weights copy in float for optim update? 0|1
    bool init_state;   // set to true if master weights need to be initialized
    int gelu_fusion; // fuse gelu via cuBLASLt (0=none, 1=forward, 2=forward+backward)
    int recompute; // recompute gelu | layernorm forward during model backward? 0|1|2
    // todo - if other functions need cpu scratch buffers in the future, reuse as generic scratch?
    int* workload_indices; // encoder_backward, B*T*num_c_groups (int)
    int4* bucket_info;     // encoder_backward, B*T*num_c_groups (int4) - size for worst case
} GPT2;

void gpt2_init_common(GPT2 *model) {
    // common inits outside of the model weights
    // memory lazily initialized in forward()
    model->acts_memory = NULL;
    model->inputs = NULL;
    model->targets = NULL;
    model->accumulated_mean_loss = NULL;
    model->cpu_losses = NULL;
    // the B,T params are determined and set, fixed on first batch in forward()
    model->batch_size = 0;
    model->seq_len = 0;
    model->mean_loss = -1.0f; // -1.0f designates no loss, set at end of forward()
    model->params_memory = NULL;
    // memory lazily initialized in backward()
    model->grads_memory = NULL;
    model->workload_indices = NULL; // on cpu, for encoder_backward
    model->bucket_info = NULL; // on cpu, for encoder_backward
    // memory lazily initialized in update()
    model->m_memory = NULL;
    model->v_memory = NULL;
    model->master_weights = NULL;
    // other default settings
    model->rng_state = 13371337 + multi_gpu_config.process_rank; // used in stochastic rounding
    model->use_master_weights = 1; // safe default: do keep master weights in fp32
    model->init_state = true;
    model->recompute = 1; // good default: recompute gelu but not layernorm
    model->gelu_fusion = 0; //deviceProp.major >= 9 ? 2 : 0; // default: off for now (default must match main())
}

void gpt2_allocate_weights(GPT2 *model) {
    // fill in all the parameter tensor dimensions and types
    fill_in_parameter_sizes(model->param_elements, model->param_sizeof, model->config);
    model->num_parameters = 0;
    model->num_parameters_bytes = 0;
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        model->num_parameters += model->param_elements[i];
        model->num_parameters_bytes += model->param_elements[i] * model->param_sizeof[i];
    }
    // create memory for model parameters on the device
    assert(model->params_memory == nullptr);
    model->params_memory = malloc_and_point_parameters(&model->params, model->param_elements, model->param_sizeof);
}

void gpt2_allocate_state(GPT2 *model, int B, int T) {
    printf0("allocating %d MiB for parameter gradients\n", (int)round(model->num_parameters * sizeof(floatX) / (1024 * 1024)));
    assert(model->grads_memory == nullptr);
    model->grads_memory = malloc_and_point_parameters(&model->grads, model->param_elements, model->param_sizeof);

    // record the current B,T as well
    model->batch_size = B;
    model->seq_len = T;

    // allocate the space
    fill_in_activation_sizes(&model->acts, model->acts_specs, B, T, model->config, model->recompute);
    model->acts_memory = malloc_and_point_activations(model->acts_specs);
    // also create memory for caching inputs and targets
    cudaCheck(hipMalloc((void**)&model->inputs, B * T * sizeof(int)));
    cudaCheck(hipMalloc((void**)&model->targets, B * T * sizeof(int)));
    cudaCheck(hipMalloc(((void**)&model->accumulated_mean_loss), sizeof(float)));
    cudaCheck(hipHostMalloc((void**)&model->cpu_losses, B * T * sizeof(float)));

    // initialise cpu scratch buffers for encoder backward
    size_t num_c_groups = CEIL_DIV(model->config.channels, (WARP_SIZE * x128::size));
    assert((size_t)(model->batch_size * model->seq_len) * num_c_groups < (1ULL<<31ULL)); // todo - maybe an issue for llama3-400B(?)
    model->workload_indices = (int*)mallocCheck(sizeof(int) * model->batch_size * model->seq_len * num_c_groups);
    model->bucket_info = (int4*)mallocCheck(sizeof(int4) * model->batch_size * model->seq_len * num_c_groups);

    // cudaMallocConditionallyManaged can fall back to hipMallocManaged if not enough memory on device
    // and returns a status code of 1 if it had to fall back, in that case we want to print warning.
    int memory_status = 0;

    // we will now init the optimizer states and master weights
    // this is usually a substantial amount of memory allocation right here.
    size_t shard_num_parameters = multi_gpu_config.shard_num_parameters; // num parameters we are responsible for
    printf0("allocating %zu MiB for AdamW optimizer state m\n", (shard_num_parameters * sizeof(float)) >> 20);
    printf0("allocating %zu MiB for AdamW optimizer state v\n", (shard_num_parameters * sizeof(float)) >> 20);
    assert(model->m_memory == nullptr);
    assert(model->v_memory == nullptr);
    memory_status |= cudaMallocConditionallyManaged((void**)&model->m_memory, shard_num_parameters * sizeof(float));
    memory_status |= cudaMallocConditionallyManaged((void**)&model->v_memory, shard_num_parameters * sizeof(float));

    if (model->use_master_weights == 1) {
        assert(model->master_weights == nullptr);
        printf0("allocating %zu MiB for master copy of params\n", (shard_num_parameters * sizeof(float)) >> 20);
        memory_status |= cudaMallocConditionallyManaged((void**) &model->master_weights, shard_num_parameters * sizeof(float));
    }

    // report on mixed memory allocation status (re-using our float reduce function, bit awk ok)
    int reduced_memory_status = (int) multi_gpu_cpu_float_sum((float)memory_status, &multi_gpu_config);
    if (reduced_memory_status >= 1) {
        printf0("WARNING: Fell back to hipMallocManaged when initializing m,v,master_weights on %d GPUs\n", reduced_memory_status);
        printf0("         Prevents an OOM, but code may run much slower due to device <-> host memory movement\n");
    }
    // report on device memory usage
    size_t free, total;
    cudaCheck(hipMemGetInfo(&free, &total));
    printf0("device memory usage: %zd MiB / %zd MiB\n", (total-free) / 1024 / 1024, total / 1024 / 1024);
    // give an estimate of the maximum batch size
    size_t bytes_per_sequence = 0;
    for (size_t i = 0; i < NUM_ACTIVATION_TENSORS; i++) {
        bytes_per_sequence += model->acts_specs[i].size * sizeof_dtype(model->acts_specs[i].type) / B;
    }
    printf0("memory per sequence: %zu MiB\n", bytes_per_sequence / 1024 / 1024);
    printf0(" -> estimated maximum batch size: %zu\n", B + free / bytes_per_sequence);
}

void gpt2_write_to_checkpoint(GPT2 *model, const char* checkpoint_path) {
    // write the model to a checkpoint file
    printf0("Writing model to %s\n", checkpoint_path);
    FILE *model_file = fopenCheck(checkpoint_path, "wb");
    // write the header first
    int model_header[256];
    memset(model_header, 0, sizeof(model_header));
    model_header[0] = 20240326; // magic number
    assert(PRECISION_MODE == PRECISION_FP32 || PRECISION_MODE == PRECISION_BF16);
    model_header[1] = PRECISION_MODE == PRECISION_FP32 ? 3 : 5; // version
    model_header[2] = model->config.max_seq_len;
    model_header[3] = model->config.vocab_size;
    model_header[4] = model->config.num_layers;
    model_header[5] = model->config.num_heads;
    model_header[6] = model->config.channels;
    model_header[7] = model->config.padded_vocab_size;
    fwriteCheck(model_header, sizeof(int), 256, model_file);
    // write the parameters
    device_to_file(model_file, model->params_memory, model->num_parameters_bytes,
                   IO_BUF_SIZE, main_stream);
    // close file, we're done
    fcloseCheck(model_file);
}

void gpt2_build_from_checkpoint(GPT2 *model, const char* checkpoint_path, bool weight_init=true) {
    // If weight_init is true, we will load the weights from this checkpoint .bin file
    // We sometimes want this to be false, if we are going to initialize these weights from
    // the master weights that are instead stored in the state .bin file.
    // In that case, this function mostly loads the model hyperparameters from the header.

    if (PRECISION_MODE == PRECISION_FP16) {
        // TODO for later perhaps, would require us dynamically converting the
        // model weights from fp32 to fp16 online, here in this function, or writing
        // the fp16 weights directly from Python, which we only do for fp32/bf16 atm.
        fprintf(stderr, "build_from_checkpoint() does not support fp16 right now.\n");
        exit(EXIT_FAILURE);
    }

    // read in model from a checkpoint file
    FILE *model_file = fopenCheck(checkpoint_path, "rb");
    int model_header[256];
    freadCheck(model_header, sizeof(int), 256, model_file);
    if (model_header[0] != 20240326) { printf("Bad magic model file\n"); exit(EXIT_FAILURE); }
    int version = model_header[1];
    if (!(version == 3 || version == 5)) {
        // 3 = fp32, padded vocab
        // 5 = bf16, padded vocab, layernorms also in bf16
        fprintf(stderr, "Bad version in model file\n");
        fprintf(stderr, "---> HINT: try to re-run `python train_gpt2.py`\n");
        exit(EXIT_FAILURE);
    }

    // check if the precision mode of the checkpoing matches the model precision
    if (weight_init) {
        if (PRECISION_MODE == PRECISION_BF16 && version != 5) {
            fprintf(stderr, "Precision is configured as BF16 but model at %s is not.\n", checkpoint_path);
            fprintf(stderr, "---> HINT: are you sure you're loading a _bf16.bin file?\n");
            exit(EXIT_FAILURE);
        }
        if (PRECISION_MODE == PRECISION_FP32 && version != 3) {
            fprintf(stderr, "Precision is configured as FP32 but model at %s is not.\n", checkpoint_path);
            fprintf(stderr, "---> HINT: to turn on FP32 you have to compile like: `make train_gpt2cu PRECISION=FP32`\n");
            fprintf(stderr, "---> HINT: are you sure you're loading a .bin file without any _bf16 in the name?\n");
            exit(EXIT_FAILURE);
        }
    }

    // read in hyperparameters
    model->config.max_seq_len = model_header[2];
    model->config.vocab_size = model_header[3];
    model->config.num_layers = model_header[4];
    model->config.num_heads = model_header[5];
    model->config.channels = model_header[6];
    model->config.padded_vocab_size = model_header[7];

    // allocate memory for the model parameters
    gpt2_allocate_weights(model);

    // read in the parameters if weight_init is true
    if (weight_init) {
        assert(model->params_memory != NULL);
        file_to_device(model->params_memory, model_file, model->num_parameters_bytes, IO_BUF_SIZE, main_stream);
    }
    fcloseCheck(model_file);

    // only return from this function once we are certain the params are ready on the GPU
    cudaCheck(hipDeviceSynchronize());
}

void gpt2_set_hyperparameters(GPT2Config* config, const char* depth_str) {
    int depth = atoi(depth_str);
    assert(depth > 0); // atoi returns 0 if not a number
    int channels, num_heads;
    if      (depth == 6)  { channels = 384; num_heads = 6; }   // (unofficial) gpt2-tiny (30M)
    else if (depth == 12) { channels = 768; num_heads = 12; }  // gpt2 (124M)
    else if (depth == 24) { channels = 1024; num_heads = 16; } // gpt2-medium (350M)
    else if (depth == 36) { channels = 1280; num_heads = 20; } // gpt2-large (774M)
    else if (depth == 48) { channels = 1600; num_heads = 25; } // gpt2-xl (1558M)
    else if (depth == 60) { channels = 1920; num_heads = 30; } // (unofficial) 2.7B
    else if (depth == 72) { channels = 2880; num_heads = 30; } // (unofficial) 7.3B
    else if (depth == 84) { channels = 3456; num_heads = 36; } // (unofficial) 12.2B
    else { fprintf(stderr, "Unsupported GPT-2 depth: %d\n", depth); exit(EXIT_FAILURE); }
    config->num_layers = depth;
    config->channels = channels;
    config->num_heads = num_heads;
    config->max_seq_len = 1024;
}

void gpt3_set_hyperparameters(GPT2Config* config, const char* channels_str) {
    // we use channels instead of depth for GPT-3 because GPT-3 model depths are not one-to-one
    // note that our models are not necessarily identical to GPT-3 because
    // we use dense attention, not the alternating dense/banded attention of GPT-3
    int channels = atoi(channels_str);
    assert(channels > 0); // atoi returns 0 if not a number
    int depth, head_size;
    if      (channels == 384)   { depth = 6;  head_size = 64; }  // (unofficial) gpt3-tiny (31M)
    else if (channels == 768)   { depth = 12; head_size = 64; }  // gpt3-small (125M)
    else if (channels == 1024)  { depth = 24; head_size = 64; }  // gpt3-medium (350M)
    else if (channels == 1536)  { depth = 24; head_size = 96; }  // gpt3-large (760M)
    else if (channels == 2048)  { depth = 24; head_size = 128; } // gpt3-xl (1.3B) [heads fixed]
    else if (channels == 2560)  { depth = 32; head_size = 80; }  // gpt3-2.7B
    else if (channels == 4096)  { depth = 32; head_size = 128; } // gpt3-6.7B
    else if (channels == 5140)  { depth = 40; head_size = 128; } // gpt3-13B
    else if (channels == 12288) { depth = 96; head_size = 128; } // gpt3 (175B)
    else { fprintf(stderr, "Unsupported GPT-3 channels: %d\n", channels); exit(EXIT_FAILURE); }
    assert(channels % head_size == 0);
    config->num_layers = depth;
    config->channels = channels;
    config->num_heads = channels / head_size;
    config->max_seq_len = 2048; // NOTE: GPT-3 uses context length of 2048 tokens, up from 1024 in GPT-2
}

void gpt_build_from_descriptor(GPT2 *model, const char* descriptor) {
    // The model descriptor can be:
    // - legacy format "dX", where X is number, e.g. "d12". This creates GPT-2 model with 12 layers.
    // - new explicit format "gpt2:dX", same as above, e.g. "gpt2:d48" for GPT-2 with 48 layers.
    // - "gpt3:cX", where X is now the channel count, e.g. "gpt3:c768" is the smallest GPT-3 model.

    // check the valid prexies and dispatch to the right setup function
    assert(descriptor != NULL);
    size_t len = strlen(descriptor);
    if (len > 1 && descriptor[0] == 'd') {
        gpt2_set_hyperparameters(&model->config, descriptor + 1); // pass along the depth str without the 'd'
    } else if (len > 6 && strncmp(descriptor, "gpt2:d", 6) == 0) {
        gpt2_set_hyperparameters(&model->config, descriptor + 6); // pass along the depth str without the 'gpt2:d'
    } else if (len > 6 && strncmp(descriptor, "gpt3:c", 6) == 0) {
        gpt3_set_hyperparameters(&model->config, descriptor + 6); // pass along the channels str without the 'gpt3:c'
    } else {
        fprintf(stderr, "Unsupported model descriptor: %s\n", descriptor); exit(EXIT_FAILURE);
    }

    // both GPT-2 and GPT-3 use the same tokenizer with 50257 tokens
    model->config.vocab_size = 50257;
    model->config.padded_vocab_size = 50304; // padded to 128 for CUDA kernel efficiency

    gpt2_allocate_weights(model);

    // allocate and random init the memory for all the parameters with GPT-2 schema
    // weights ~N(0, 0.02), biases 0, c_proj weights ~N(0, 0.02/(2*L)**0.5)
    // NOTE: assuming all parameters are of the type floatX, could be relaxed later
    mt19937_state init_rng;
    manual_seed(&init_rng, 42);
    floatX* params_memory_cpu = (floatX*)mallocCheck(model->num_parameters_bytes);
    memset(params_memory_cpu, 0, model->num_parameters_bytes);
    // fill in all the weights with random values
    float residual_scale = 1.0f / sqrtf(2.0f * model->config.num_layers);
    // we have to init all these tensors exactly in the order that PyTorch initializes them
    // so that we can match them up and get correctness and exactly the same initial conditions
    size_t L = model->config.num_layers;
    size_t offset = 0;
    for (int l = 0; l < L; l++) {
        offset = 0;
        for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
            // the layernorm parameters are all initialized to 1
            if (l == 0 && (i == 2 || i == 8 || i == 14)) { // only at l = 0 to init these just once
                for (size_t j = 0; j < model->param_elements[i]; j++) {
                    params_memory_cpu[offset + j] = 1.0f;
                }
            }
            // weights tensors are handled here
            if ((l == 0 && (i == 0 || i == 1)) // only at l = 0, init the wte and wpe tensors
              || i == 4 || i == 6 || i == 10 || i == 12) {
                size_t n = model->param_elements[i];
                size_t layer_offset = 0;
                if (i == 0) {
                    // for wte tensor (padded vocab) override to init V instead of Vp rows
                    n = model->config.vocab_size * model->config.channels;
                }
                if (i == 4 || i == 6 || i == 10 || i == 12) {
                    // weight tensors, we are only initializing layer l
                    assert(n % L == 0);
                    n = n / L;
                    layer_offset = l * n;
                }
                // in GPT-2, the projections back into the residual stream are additionally
                // scaled by 1/sqrt(2*L) for training stability
                float scale = (i == 6 || i == 12) ? 0.02f * residual_scale : 0.02f;
                // okay let's draw the random numbers and write them
                float *fp32_buffer = (float*)mallocCheck(n * sizeof(float));
                normal_(fp32_buffer, n, 0.0f, scale, &init_rng);
                for (size_t j = 0; j < n; j++) {
                    params_memory_cpu[offset + layer_offset + j] = (floatX)fp32_buffer[j];
                }
                free(fp32_buffer);
            }
            offset += model->param_elements[i];
        }
    }

    // copy them to GPU
    cudaCheck(hipMemcpy(model->params_memory, params_memory_cpu, model->num_parameters_bytes, hipMemcpyHostToDevice));
    free(params_memory_cpu);
}

// propagate inputs through the network to produce logits.
// right now, this function is fully synchronous with the host
void gpt2_forward(GPT2 *model, const int* inputs, size_t B, size_t T) {
    NVTX_RANGE_FN();
    // we must be careful and use size_t instead of int, otherwise
    // we could overflow int. E.g. l * B * NH * T * T overflows int at B 16.

    // ensure the model was initialized or error out
    if (model->params_memory == NULL) {
        printf("Error: model was not initialized properly.\n");
        exit(EXIT_FAILURE);
    }

    // convenience parameters
    const size_t V = model->config.vocab_size;
    const size_t Vp = model->config.padded_vocab_size;
    const size_t L = model->config.num_layers;
    const size_t NH = model->config.num_heads;
    const size_t C = model->config.channels;

    // validate B,T are not larger than the values used at initialisation
    // (smaller B,T are okay for inference only)
    if (B > model->batch_size || T > model->seq_len) {
        printf("Model: B=%d T=%d, Desired: B=%d T=%d\n", model->batch_size, model->seq_len, (int)B, (int)T);
        exit(EXIT_FAILURE);
    }

    // copy inputs/targets to the model
    cudaCheck(hipMemcpy(model->inputs, inputs, B * T * sizeof(int), hipMemcpyHostToDevice));
    // validate inputs, all indices must be in the range [0, V)
    // we can do this while the copies are already underway
    tokenCheck(inputs, B*T, V);

    // forward pass
    ParameterTensors params = model->params; // for brevity
    ActivationTensors acts = model->acts;
    encoder_forward(acts.encoded, model->inputs, params.wte, params.wpe, B, T, C, main_stream); // encoding goes into residual[0]

    // first layernorm isn't fused
    layernorm_forward((model->recompute < 2) ? acts.ln1 : acts.lnf, acts.ln1_mean, acts.ln1_rstd, acts.encoded, params.ln1w, params.ln1b, B, T, C, main_stream);

    for (int l = 0; l < L; l++) {
        NvtxRange layer_range("Layer", l);

        floatX* residual = l == 0 ? acts.encoded : acts.residual3 + (l-1) * B * T * C;

        // get the pointers of the weights for this layer
        floatX* l_qkvw = params.qkvw + l * 3*C * C;
        floatX* l_qkvb = params.qkvb + l * 3*C;
        floatX* l_attprojw = params.attprojw + l * C * C;
        floatX* l_attprojb = params.attprojb + l * C;
        floatX* l_ln2w = params.ln2w + l * C;
        floatX* l_ln2b = params.ln2b + l * C;
        floatX* l_fcw = params.fcw + l * 4*C * C;
        floatX* l_fcb = params.fcb + l * 4*C;
        floatX* l_fcprojw = params.fcprojw + l * C * 4*C;
        floatX* l_fcprojb = params.fcprojb + l * C;

        // get the pointers of the activations for this layer
        floatX* l_ln1 = (model->recompute < 2) ? acts.ln1 + l * B * T * C : acts.lnf;
        floatX* l_qkvr = acts.qkvr + l * B * T * 3*C;
        floatX* l_atty = acts.atty + l * B * T * C;
        floatX* l_residual2 = acts.residual2 + l * B * T * C;
        floatX* l_ln2 = (model->recompute < 2) ? acts.ln2 + l * B * T * C : acts.lnf;
        float* l_ln2_mean = acts.ln2_mean + l * B * T;
        float* l_ln2_rstd = acts.ln2_rstd + l * B * T;
        floatX* l_fch = acts.fch + l * B * T * 4*C;
        // reuse the same activation buffer at each layer, as we'll re-compute the gelu during backward
        // very useful because we dramatically reduce VRAM usage, and may be able to fit larger batch size
        floatX* l_fch_gelu = (model->recompute < 1) ? acts.fch_gelu + l * B * T * 4*C : acts.fch_gelu;
        floatX* l_residual3 = acts.residual3 + l * B * T * C;
        floatX* scratch = (floatX*)acts.output; // used for non-cudnn attention, fcproj, attproj, etc.

        // now do the forward pass
        #ifdef ENABLE_CUDNN
        float* l_att = (float*)acts.att + l * B * NH * T; // cuDNN needs a smaller FP32 tensor
        matmul_forward_cublaslt(l_qkvr, l_ln1, l_qkvw, l_qkvb, B, T, C, 3*C, main_stream);
        attention_forward_cudnn(l_atty, (float*)l_att, l_qkvr, B, T, NH, C, main_stream);
        #else
        floatX* l_att = acts.att + l * B * NH * T * T;
        if (T != model->seq_len) { // unused parts of attention buffer must be zeroed (T-dependent)
            cudaCheck(hipMemset(l_att, 0, B * NH * T * T * sizeof(floatX)));
        }
        // these are only needed as scratchpads for the forward pass, but
        // need not be stored for backward
        matmul_forward_cublaslt(scratch, l_ln1, l_qkvw, l_qkvb, B, T, C, 3*C, main_stream);
        attention_forward(l_atty, l_qkvr, l_att, scratch, B, T, C, NH, main_stream);
        #endif

        matmul_forward_cublaslt(scratch, l_atty, l_attprojw, l_attprojb, B, T, C, C, main_stream);
        fused_residual_forward5(l_residual2, l_ln2, l_ln2_mean, l_ln2_rstd, residual, scratch, l_ln2w, l_ln2b, B*T, C, main_stream);
        matmul_forward_cublaslt(l_fch_gelu, l_ln2, l_fcw, l_fcb, B, T, C, 4*C, main_stream, l_fch, model->gelu_fusion);
        matmul_forward_cublaslt(scratch, l_fch_gelu, l_fcprojw, l_fcprojb, B, T, 4*C, C, main_stream);
        // OK, fusion across blocks.
        if(l+1 != L) {
            floatX* l_ln1 = (model->recompute < 2) ? acts.ln1 + (l + 1) * B * T * C : acts.lnf;
            float* l_ln1_mean = acts.ln1_mean + (l + 1) * B * T;
            float* l_ln1_rstd = acts.ln1_rstd + (l + 1) * B * T;
            const floatX* l_ln1w = params.ln1w + (l + 1) * C;
            const floatX* l_ln1b = params.ln1b + (l + 1) * C;
            fused_residual_forward5(l_residual3, l_ln1, l_ln1_mean, l_ln1_rstd, l_residual2, scratch, l_ln1w, l_ln1b,
                                    B * T, C, main_stream);
        } else {
            fused_residual_forward5(l_residual3, acts.lnf, acts.lnf_mean, acts.lnf_rstd, l_residual2, scratch,
                                    params.lnfw, params.lnfb,
                                    B * T, C, main_stream);
        }
    }

    matmul_forward_cublaslt(acts.output, acts.lnf, params.wte, NULL, B, T, C, Vp, main_stream);
    cudaCheck(hipDeviceSynchronize());
}


// Forwards both the model and the loss and is used for validation splits and evals.
// In particular it populates cpu_losses with loss at each token.
// Some of the evals (e.g. HellaSwag) require the per-token losses, which are produced here.
float gpt2_validate(GPT2 *model, const int* inputs, const int* targets, size_t B, size_t T) {
    assert(targets != NULL);
    // forward the model itself
    gpt2_forward(model, inputs, B, T);
    // convenience shortcuts, size_t instead of int so that pointer arithmetics don't overflow
    const size_t V = model->config.vocab_size;
    const size_t Vp = model->config.padded_vocab_size;

    NvtxRange classifier_and_loss_range("classifier_and_loss");
    ActivationTensors acts = model->acts;
    float mean_loss = 0.0f;
    // fused classifier: does the forward pass and first part of the backward pass
    const float dloss = 1.0f / (B * T); // results in the uniform average loss over all elements
    // note: we don't need to generate dlogits here
    cudaCheck(hipMemset(acts.losses, 0, B*T*sizeof(float)));
    cudaCheck(hipMemcpy(model->targets, targets, B * T * sizeof(int), hipMemcpyHostToDevice));
    tokenCheck(targets, B*T, V); // while the memcpy is underway, validate the targets
    fused_classifier(acts.output, acts.losses, dloss, model->targets, B, T, V, Vp, False, main_stream);
    cudaCheck(hipMemcpy(model->cpu_losses, acts.losses, B * T * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < B*T; i++) {
        mean_loss += model->cpu_losses[i];
    }
    mean_loss /= B*T;
    cudaCheck(hipDeviceSynchronize());
    return mean_loss;
}

void gpt2_backward_and_reduce(GPT2 *model, int* inputs, const int* targets, int grad_accum_steps, int micro_step) {
    if(model->grads_memory == nullptr) {
        fprintf(stderr, "Need to allocate gradients before backward");
        exit(EXIT_FAILURE);
    }
    NVTX_RANGE_FN();
    bool last_step = micro_step == grad_accum_steps - 1;
    // on the first micro-step zero the gradients, as we're about to += accumulate into them
    if (micro_step == 0) {
        // there are currently two state vars during the gradient accumulation inner loop:
        // 1) the losses accumulate += into acts.losses, reset here
        // 2) the gradients accumulate += into grads_memory, reset here
        cudaCheck(hipMemsetAsync(model->acts.losses, 0, model->batch_size * model->seq_len * sizeof(float), main_stream));
        cudaCheck(hipMemsetAsync(model->grads_memory, 0, model->num_parameters * sizeof(floatX), main_stream));
    }

    // convenience shortcuts, size_t instead of int so that pointer arithmetics don't overflow
    const size_t B = model->batch_size;
    const size_t T = model->seq_len;
    const size_t V = model->config.vocab_size;
    const size_t Vp = model->config.padded_vocab_size;
    const size_t L = model->config.num_layers;
    const size_t NH = model->config.num_heads;
    const size_t C = model->config.channels;

    ParameterTensors params = model->params; // for brevity
    ParameterTensors grads = model->grads;
    ActivationTensors acts = model->acts;

    // accumulate the losses inside acts.losses, and kick off the backward pass inside the fused classifier
    NvtxRange classifier_and_loss_range("classifier_and_loss");
    const float dloss = 1.0f / (float)(B * T * grad_accum_steps); // results in the uniform average loss over all elements
    cudaCheck(hipMemcpy(model->targets, targets, B * T * sizeof(int), hipMemcpyHostToDevice));
    tokenCheck(targets, B*T, V);
    fused_classifier(acts.output, acts.losses, dloss, model->targets, B, T, V, Vp, True, main_stream);

    // backward pass: go in the reverse order of the forward pass, and call backward() functions

    // reset residual stream gradients (put here to work with gradient accumulation)
    floatX* dresidual = (floatX*)model->acts.scratch_btc; // the main buffer holding the gradient in the backward pass
    cudaCheck(hipMemset(dresidual, 0, B * T * C * sizeof(floatX)));

    // re-use the output buffer of the forward pass as a scratchpad during backward pass
    float* scratchF = (float*)acts.output;
    floatX* scratchX = (floatX*)acts.output;

    // we kick off the chain rule by filling in dlosses with 1.0f/(B*T)
    // this was done in the fused classifier kernel as last step of forward pass
    // technically that is a small, inline backward() pass of calculating
    // total, final loss as the mean over all losses over all (B,T) positions in the batch
    // next: backward the classifier matmul
    matmul_backward(model->acts.scratch_bt4c, grads.wte, NULL, acts.output, acts.lnf, params.wte, NULL, B, T, C, Vp, main_stream);
    // backward the final layernorm
    floatX* residual = acts.residual3 + (L-1) * B * T * C; // last residual is in residual3
    layernorm_backward(dresidual, grads.lnfw, grads.lnfb, scratchF, model->acts.scratch_bt4c, residual, params.lnfw, acts.lnf_mean, acts.lnf_rstd, B, T, C, main_stream);

    // from this point on, we no longer need the values stored in the last residual, so we can reuse that memory as generic
    // scratch for backward computations
    floatX* dl_btc = residual;

    // now backward all the layers
    for (int l = L-1; l >= 0; l--) {
        NvtxRange layer_range("Layer", l);

        residual = l == 0 ? acts.encoded : acts.residual3 + (l-1) * B * T * C;

        // get the pointers of the weights for this layer
        floatX* l_ln1w = params.ln1w + l * C;
        floatX* l_ln1b = params.ln1b + l * C;
        floatX* l_qkvw = params.qkvw + l * 3*C * C;
        floatX* l_attprojw = params.attprojw + l * C * C;
        floatX* l_ln2w = params.ln2w + l * C;
        floatX* l_ln2b = params.ln2b + l * C;
        floatX* l_fcw = params.fcw + l * 4*C * C;
        floatX* l_fcprojw = params.fcprojw + l * C * 4*C;
        // get the pointers of the gradients of the weights for this layer
        floatX* dl_ln1w = grads.ln1w + l * C;
        floatX* dl_ln1b = grads.ln1b + l * C;
        floatX* dl_qkvw = grads.qkvw + l * 3*C * C;
        floatX* dl_qkvb = grads.qkvb + l * 3*C;
        floatX* dl_attprojw = grads.attprojw + l * C * C;
        floatX* dl_attprojb = grads.attprojb + l * C;
        floatX* dl_ln2w = grads.ln2w + l * C;
        floatX* dl_ln2b = grads.ln2b + l * C;
        floatX* dl_fcw = grads.fcw + l * 4*C * C;
        floatX* dl_fcb = grads.fcb + l * 4*C;
        floatX* dl_fcprojw = grads.fcprojw + l * C * 4*C;
        floatX* dl_fcprojb = grads.fcprojb + l * C;
        // get the pointers of the activations for this layer
        floatX* l_ln1 = (model->recompute < 2) ? acts.ln1 + l * B * T * C : acts.lnf;
        float* l_ln1_mean = acts.ln1_mean + l * B * T;
        float* l_ln1_rstd = acts.ln1_rstd + l * B * T;
        floatX* l_qkvr = acts.qkvr + l * B * T * 3*C;
        floatX* l_atty = acts.atty + l * B * T * C;
        floatX* l_residual2 = acts.residual2 + l * B * T * C;
        floatX* l_ln2 = (model->recompute < 2) ? acts.ln2 + l * B * T * C : acts.lnf;
        float* l_ln2_mean = acts.ln2_mean + l * B * T;
        float* l_ln2_rstd = acts.ln2_rstd + l * B * T;
        floatX* l_fch_pre_gelu = acts.fch + l * B * T * 4*C;
        floatX* l_fch_gelu = (model->recompute < 1) ? acts.fch_gelu + l * B * T * 4*C : acts.fch_gelu;
        // get the pointers of the gradients of the activations for this layer
        // notice that there is no l *, because we just have a single copy, and keep
        // re-using this memory in every Transformer block as we calculate backward pass

        floatX* dl_bt4c = (floatX*)model->acts.scratch_bt4c;

        // start the backward pass for this layer
        if(model->recompute >= 1) {
            // recompute >= 1 means we recompute gelu. in this case,
            // l_fch_gelu is just a buffer, so re-compute the gelu from l_fch here
            gelu_forward(l_fch_gelu, l_fch_pre_gelu, B*T*4*C, main_stream);
        }
        matmul_backward(dl_bt4c, dl_fcprojw, dl_fcprojb, dresidual, l_fch_gelu, l_fcprojw, scratchF, B, T, 4*C, C, main_stream, l_fch_pre_gelu, model->gelu_fusion);
        if(model->recompute >= 2) {
            // same as gelu above, l_ln1 and l_ln2 are just buffers if recompute >= 2, recompute them here on demand
            layernorm_forward(l_ln2, l_ln2_mean, l_ln2_rstd, l_residual2, l_ln2w, l_ln2b, B, T, C, main_stream);
        }
        matmul_backward(dl_btc, dl_fcw, dl_fcb, dl_bt4c, l_ln2, l_fcw, scratchF, B, T, C, 4 * C, main_stream);
        // layernorm backward does += to the dresidual, so it correctly accumulates grad from the MLP block above
        layernorm_backward(dresidual, dl_ln2w, dl_ln2b, scratchF, dl_btc, l_residual2, l_ln2w, l_ln2_mean, l_ln2_rstd, B, T, C, main_stream);
        matmul_backward(dl_btc, dl_attprojw, dl_attprojb, dresidual, l_atty, l_attprojw, scratchF, B, T, C, C, main_stream);

        #ifdef ENABLE_CUDNN
        float* l_att = (float*)acts.att + l * B * NH * T; // cuDNN needs a smaller FP32 tensor
        attention_backward_cudnn(dl_bt4c, dl_btc, l_qkvr, l_atty, (float*)l_att, B, T, NH, C, main_stream);
        #else
        floatX* l_att = acts.att + l * B * NH * T * T;
        // we need B x T x (4)C buffers. l_atty and l_fch aren't needed anymore at this point, so reuse their memory
        floatX* buffer_a = l_atty;
        floatX* buffer_b = l_fch_pre_gelu;        // this is B x T x 4C, so even larger than what we need
        attention_backward(dl_bt4c, buffer_b, scratchX, buffer_a, dl_btc, l_qkvr, l_att, B, T, C, NH, main_stream);
        #endif
        if(model->recompute >= 2) {
            layernorm_forward(l_ln1, l_ln1_mean, l_ln1_rstd, residual, l_ln1w, l_ln1b, B, T, C, main_stream);
        }
        // QKV parameter gradients
        matmul_backward(dl_btc, dl_qkvw, dl_qkvb, dl_bt4c, l_ln1, l_qkvw, scratchF, B, T, C, 3 * C, main_stream);
        // layernorm backward does += to dresidual, so it correctly accumulates gradient for the Attention block above
        layernorm_backward(dresidual, dl_ln1w, dl_ln1b, scratchF, dl_btc, residual, l_ln1w, l_ln1_mean, l_ln1_rstd, B, T, C, main_stream);

        // Accumulate gradients from this layer in a background stream.
        if(last_step) {
            floatX* const pointers[] = {
                dl_ln1w, dl_ln1b,
                dl_qkvw, dl_qkvb,
                dl_attprojw, dl_attprojb,
                dl_ln2w, dl_ln2b,
                dl_fcw, dl_fcb,
                dl_fcprojw, dl_fcprojb
            };
            const size_t nelem[] = {
                C, C,
                3 * C * C, 3 * C,
                C * C, C,
                C, C,
                4 * C * C, 4 * C,
                C * 4 * C, C
            };
            multi_gpu_async_reduce_gradient(pointers, nelem, &multi_gpu_config, main_stream);
        }
    }
    encoder_backward(grads.wte, grads.wpe, scratchX, model->workload_indices, model->bucket_info,
                     dresidual, model->inputs, inputs, B, T, C, random_u32(&model->rng_state), main_stream);

    // Aggregate all gradients that are not part of the transformer blocks
    if(last_step) {
        // reduce all the losses within the current GPU (across all microsteps)
        global_sum_deterministic(model->accumulated_mean_loss, acts.losses, B*T, main_stream);
        // reduce loss across GPUs to a single, final float across all microsteps and GPUs
        #if MULTI_GPU
        ncclCheck(ncclAllReduce(model->accumulated_mean_loss, model->accumulated_mean_loss, sizeof(float), ncclFloat, ncclAvg, multi_gpu_config.nccl_comm, main_stream));
        #endif
        cudaCheck(hipMemcpyAsync(&model->mean_loss, model->accumulated_mean_loss, sizeof(float), hipMemcpyDeviceToHost, main_stream));
        // reduce the gradients for non-transformer block parameters
        floatX* const pointers[] = {grads.wte, grads.wpe, grads.lnfw, grads.lnfb};
        const size_t nelem[] = {Vp * C, T * C, C, C};
        multi_gpu_async_reduce_gradient(pointers, nelem, &multi_gpu_config, main_stream);
    }

    cudaCheck(hipDeviceSynchronize());
    if(last_step) {
        model->mean_loss /= B*T*grad_accum_steps;
    } else {
        model->mean_loss = -1.f; // no loss available yet
    }
}

// Gets the offset of a specific tensor for a specific layer in the GPT2 model
// layer_id is ignored for weights that are not part of a transformer block
ShardInfo gpt2_get_tensor_at_layer(const GPT2 *model, int layer_id, int param_tensor_id) {
    // first offset our way to the parameter tensor start
    ptrdiff_t offset = 0;
    for (int i = 0; i < param_tensor_id; i++) {
        offset += (ptrdiff_t)model->param_elements[i];
    }
    size_t size = model->param_elements[param_tensor_id] ;
    // if we are in the transformer block, we need to additionally offset by the layer id
    if(2 <= param_tensor_id && param_tensor_id <= 13) {
        size /= model->config.num_layers;
        offset += (ptrdiff_t)(layer_id * size);
    }
    return {offset, size};
}

float gpt2_calculate_grad_norm(GPT2 *model, MultiGpuConfig* multi_gpu_config) {
    NVTX_RANGE_FN();
    floatX* grads_memory = (floatX*)model->grads_memory;

    // repurposing this buffer (which isn't needed now) to write grad norm into it
    float* grad_norm_squared = (float*)model->acts.output;
    float grad_norm_squared_cpu = 0.0f;

    int num_slices[2] = {1, model->config.num_layers};
    int max_num_block_sums = get_max_num_block_sums(num_slices, 2);
    if (multi_gpu_config->zero_stage == 1) {
        // because of the ncclReduceScatter() in backward,
        // grads_memory only contains the averaged gradients at the local shards,
        // so we only calculate the grad norm at the grads_memory belonging to the local shards
        for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
            ShardInfo tensor = gpt2_get_tensor_at_layer(model, 0, i);
            ShardInfo shard = multi_gpu_get_shard_offset(tensor.size, multi_gpu_config, 1);
            ptrdiff_t offset = tensor.offset + shard.offset;
            bool is_first_pass = (i == 0);
            if((i < 2 || i > 13)) {
                global_norm_squared(grad_norm_squared, grads_memory + offset, shard.size, 0, 1,
                                    max_num_block_sums, is_first_pass, main_stream);
            } else {
                global_norm_squared(grad_norm_squared, grads_memory + offset, shard.size, tensor.size, model->config.num_layers,
                                    max_num_block_sums, is_first_pass, main_stream);
            }
        }
        global_sum_deterministic(grad_norm_squared, grad_norm_squared, max_num_block_sums, main_stream);
#if MULTI_GPU
        // further sum the (partial) squared norm across all GPUs
        ncclCheck(ncclAllReduce(grad_norm_squared, grad_norm_squared, sizeof(float), ncclFloat, ncclSum, multi_gpu_config->nccl_comm, main_stream));
#endif
    } else {
        // in regular DDP, backward has averaged the gradients across all GPUs
        // so each GPU can compute the squared norm over the whole grad vector, with no added comms needed
        global_norm_squared(grad_norm_squared, grads_memory, model->num_parameters, 0, 1, max_num_block_sums, true, main_stream);
        global_sum_deterministic(grad_norm_squared, grad_norm_squared, max_num_block_sums, main_stream);
    }
    cudaCheck(hipMemcpy(&grad_norm_squared_cpu, grad_norm_squared, sizeof(float), hipMemcpyDeviceToHost));
    float grad_norm_cpu = sqrtf(grad_norm_squared_cpu);
    return grad_norm_cpu;
}

void gpt2_update(GPT2 *model, float learning_rate, float beta1, float beta2, float eps, float weight_decay, float grad_scale, int t,
                 MultiGpuConfig* multi_gpu_config, bool init_from_master_only=false) {
    // update the model parameters using the AdamW optimizer
    // keep in mind that optimizer sharding (ZeRO-1) assigns different parameters to different GPUs
    // so we may not be responsible for the entire parameter tensor
    // also, this function was very simple a while back but become very complex, only because we want to
    // selectively weight decay some, but not all tensors :(
    // TODO: revisit and probably refactor this entire function
    NVTX_RANGE_FN();
    if(model->grads_memory == nullptr || model->m_memory == nullptr || model->v_memory == nullptr) {
        fprintf(stderr, "Need to allocate optimizer state before update");
        exit(EXIT_FAILURE);
    }

    bool init_state = model->init_state;
    if(init_state) {
        model->init_state = false;
        NvtxRange rng("InitOpt");
        cudaCheck(hipMemset(model->m_memory, 0, multi_gpu_config->shard_num_parameters * sizeof(float)));
        cudaCheck(hipMemset(model->v_memory, 0, multi_gpu_config->shard_num_parameters * sizeof(float)));
    }

    // save RNG state at this point so we can round from master weights identically when restoring from a checkpoint
    model->rng_state_last_update = model->rng_state;

    // AdamW update
    // handle adamw for all the transformer blocks
    for (int i = 0; i < NUM_PARAMETER_TENSORS; i++) {
        // generate a unique seed for each tensor
        unsigned int seed = random_u32(&model->rng_state);

        int num_layers = model->config.num_layers;
        if((i < 2 || i > 13)) {
            num_layers = 1;
        }

        ShardInfo tensor = gpt2_get_tensor_at_layer(model, 0, i);
        ShardInfo shard = multi_gpu_get_shard_offset(tensor.size, multi_gpu_config, 1);
        ptrdiff_t local_offset_full = tensor.offset + shard.offset;
        ptrdiff_t local_offset_partial = tensor.offset / multi_gpu_config->num_processes;

        // we only want to weight decay the 2D tensors and leave all 1D tensors alone
        // in particular this also decays the embedding weights, but this is ok:
        // - the token embeddings are weight shared and participate in the final projection to logits
        // - the position embeddings actively participate at every forward/backward pass
        float wd = (i == 0 || i == 1 || i == 4 || i == 6 || i == 10 || i == 12) ? weight_decay : 0.0f;
        floatX* param_ptr = (floatX*)model->params_memory + local_offset_full;
        floatX* grad_ptr = (floatX*)model->grads_memory + local_offset_full;

        ptrdiff_t opt_state_offset = multi_gpu_config->zero_stage < 1 ?  local_offset_full : local_offset_partial;
        float* m_ptr = model->m_memory + opt_state_offset;
        float* v_ptr = model->v_memory + opt_state_offset;
        float* master_ptr = nullptr;
        if (model->master_weights != nullptr) { master_ptr = model->master_weights + opt_state_offset; }
        if(init_state && model->master_weights != nullptr ) {
            size_t grid_size = CEIL_DIV(shard.size, 512);
            copy_and_cast_kernel<<<dim3(grid_size, num_layers), 512, 0, main_stream>>>(master_ptr, param_ptr, shard.size,
                                                                     shard.size, tensor.size);
            cudaCheck(hipGetLastError());
        }

        if (init_from_master_only) {
            // when resuming training from a checkpoint with master weights (allows changing precision)
            init_from_master(param_ptr, master_ptr, shard.size, tensor.size, shard.size, num_layers, seed, main_stream);
        } else {
            // ok finally call the kernel to update the weights with AdamW
            adamw_update(param_ptr, master_ptr, grad_ptr,
                        m_ptr, v_ptr,
                        shard.size, tensor.size, tensor.size, shard.size, num_layers,
                        learning_rate,
                        beta1, beta2, t, eps, wd, grad_scale, seed, main_stream);
        }

        if (multi_gpu_config->zero_stage == 1) {
#if MULTI_GPU
            ncclCheck(ncclGroupStart());
            for(int l = 0; l < num_layers; ++l) {
                // gather updated shards of model->params_memory from each process
                ncclCheck(ncclAllGather(param_ptr + l * tensor.size,
                                        (floatX*) model->params_memory + tensor.offset + l * tensor.size,
                                        shard.size, ncclFloatX,
                                        multi_gpu_config->nccl_comm, multi_gpu_config->nccl_stream));
            }
            ncclCheck(ncclGroupEnd());
#endif
        }
    }

    cudaCheck(hipDeviceSynchronize());
}

float gpt2_estimate_mfu(GPT2 *model, int num_tokens, float dt) {
    size_t N = model->num_parameters;
    int L = model->config.num_layers;
    int C = model->config.channels;
    int T = model->seq_len;
    size_t flops_per_token = 6 * N + (size_t)6 * L * C * T;
    size_t flops_per_step = flops_per_token * num_tokens;
    float flops_achieved = (float)flops_per_step * (1.0f / dt);
    float flops_promised = get_flops_promised(deviceProp.name, PRECISION_MODE) * 1e12f;
    if(flops_promised < 0) { return -1.f; }
    float mfu = flops_achieved / flops_promised;
    return mfu;
}

void gpt2_free(GPT2 *model) {
    cudaFreeCheck(&model->params_memory);
    cudaFreeCheck(&model->grads_memory);
    cudaFreeCheck(&model->m_memory);
    cudaFreeCheck(&model->v_memory);
    cudaFreeCheck(&model->master_weights);
    cudaFreeCheck(&model->acts_memory);
    cudaFreeCheck(&model->inputs);
    cudaFreeCheck(&model->targets);
    cudaFreeCheck(&model->accumulated_mean_loss);
    cudaCheck(hipHostFree(model->cpu_losses));
    free(model->workload_indices);
    free(model->bucket_info);
}

void common_start(bool override_enable_tf32 = true, bool print_device_info = true) {
    cudaCheck(hipGetDeviceProperties(&deviceProp, multi_gpu_config.local_device_idx));
    if (print_device_info) { printf("[System]\n"); printf("Device %d: %s\n", multi_gpu_config.local_device_idx, deviceProp.name); }
    cudaCheck(hipStreamCreate(&main_stream));
    nvtxNameCudaStreamA(main_stream, "main stream");
    cublasCheck(hipblasLtCreate(&cublaslt_handle));
    cudaCheck(hipMalloc(&cublaslt_workspace, cublaslt_workspace_size));
    bool enable_tf32 = PRECISION_MODE == PRECISION_FP32 && deviceProp.major >= 8 && override_enable_tf32;
    cublas_compute = enable_tf32 ? HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
    #ifdef ENABLE_CUDNN
    create_cudnn();
    #endif
}

void common_free(GPT2 &model) {
    cudaCheck(hipStreamDestroy(main_stream));
    cudaCheck(hipFree(cublaslt_workspace));
    cublasCheck(hipblasLtDestroy(cublaslt_handle));
    #ifdef ENABLE_CUDNN
    destroy_cudnn();
    #endif
}

void save_state(const char* filename, int step, GPT2* model, DataLoader* loader) {
    printf("Writing state to %s\n", filename);
    FILE *state_file = fopenCheck(filename, "wb");
    int state_header[256];
    memset(state_header, 0, sizeof(state_header));
    state_header[0] = 20240527; state_header[1] = 1; state_header[2] = multi_gpu_config.num_processes;
    state_header[3] = multi_gpu_config.process_rank; state_header[4] = model->use_master_weights;
    state_header[5] = loader->should_shuffle; state_header[10] = step;
    *((unsigned long long*)&state_header[20]) = model->rng_state;
    *((unsigned long long*)&state_header[22]) = model->rng_state_last_update;
    *((size_t*)&state_header[30]) = loader->current_shard_idx;
    *((size_t*)&state_header[32]) = loader->current_sample_idx;
    fwriteCheck(state_header, sizeof(int), 256, state_file);
    size_t shard_num_parameters = multi_gpu_config.shard_num_parameters;
    device_to_file(state_file, model->m_memory, shard_num_parameters * sizeof(float), IO_BUF_SIZE, main_stream);
    device_to_file(state_file, model->v_memory, shard_num_parameters * sizeof(float), IO_BUF_SIZE, main_stream);
    if(model->use_master_weights) { device_to_file(state_file, model->master_weights, shard_num_parameters * sizeof(float), IO_BUF_SIZE, main_stream); }
    if (loader->should_shuffle) {
        fwriteCheck(&loader->glob_result.gl_pathc, sizeof(size_t), 1, state_file);
        fwriteCheck(loader->shard_indices, sizeof(int), loader->glob_result.gl_pathc, state_file);
        fwriteCheck(&loader->shard_num_samples, sizeof(size_t), 1, state_file);
        fwriteCheck(loader->intra_shard_indices, sizeof(int), loader->shard_num_samples, state_file);
        fwriteCheck(&loader->shuffle_rng, sizeof(mt19937_state), 1, state_file);
    }
    fcloseCheck(state_file);
}

void load_state(int* step, GPT2* model, DataLoader* loader, const char* filename) {
    FILE *state_file = fopenCheck(filename, "rb");
    int state_header[256];
    freadCheck(state_header, sizeof(int), 256, state_file);
    assert(state_header[0] == 20240527); assert(state_header[1] == 1);
    assert(state_header[2] == multi_gpu_config.num_processes); assert(state_header[3] == multi_gpu_config.process_rank);
    int use_master_weights = state_header[4]; int should_shuffle = state_header[5];
    *step = state_header[10];
    model->rng_state = *((unsigned long long*)&state_header[20]);
    model->rng_state_last_update = *((unsigned long long*)&state_header[22]);
    size_t current_shard_idx = *((size_t*)&state_header[30]);
    size_t current_sample_idx = *((size_t*)&state_header[32]);
    size_t shard_num_parameters = multi_gpu_config.shard_num_parameters;
    if(use_master_weights == 1 && !model->use_master_weights) { printf0("Warning: Master weights are present in state, but not enabled for current run."); }
    else if (use_master_weights == 0 && model->use_master_weights) { printf0("Error: Master weights requested, but not present in state file."); exit(EXIT_FAILURE); }
    model->init_state = false; assert(model->m_memory != nullptr); assert(model->v_memory != nullptr);
    file_to_device(model->m_memory, state_file, shard_num_parameters * sizeof(float), IO_BUF_SIZE, main_stream);
    file_to_device(model->v_memory, state_file, shard_num_parameters * sizeof(float), IO_BUF_SIZE, main_stream);
    if(model->use_master_weights) {
        assert(model->master_weights != nullptr);
        file_to_device(model->master_weights, state_file, shard_num_parameters * sizeof(float), IO_BUF_SIZE, main_stream);
        model->rng_state = model->rng_state_last_update;
        gpt2_update(model, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0, &multi_gpu_config, true);
        model->rng_state = *((unsigned long long*)&state_header[20]);
    }
    loader->should_shuffle = should_shuffle;
    if (should_shuffle == 1) {
        size_t glob_result_gl_pathc; freadCheck(&glob_result_gl_pathc, sizeof(size_t), 1, state_file); assert(glob_result_gl_pathc == loader->glob_result.gl_pathc);
        loader->shard_indices = (int*)mallocCheck(loader->glob_result.gl_pathc * sizeof(int));
        freadCheck(loader->shard_indices, sizeof(int), loader->glob_result.gl_pathc, state_file);
        size_t shard_num_samples; freadCheck(&shard_num_samples, sizeof(size_t), 1, state_file); assert(shard_num_samples == loader->shard_num_samples);
        loader->intra_shard_indices = (int*)mallocCheck(loader->shard_num_samples * sizeof(int));
        freadCheck(loader->intra_shard_indices, sizeof(int), loader->shard_num_samples, state_file);
        freadCheck(&loader->shuffle_rng, sizeof(mt19937_state), 1, state_file);
    }
    dataloader_resume(loader, current_shard_idx, current_sample_idx);
    fcloseCheck(state_file);
}

void write_checkpoint(const char* output_log_dir, int step, GPT2* model, DataLoader* train_loader, MultiGpuConfig* multi_gpu_config) {
    printf0("Writing checkpoint at step %d to %s\n", step, output_log_dir);
    int rank = multi_gpu_config->process_rank;
    if (rank == 0) {
        snprintf(filename_buffer, sizeof(filename_buffer), "%s/model_%08d.bin", output_log_dir, step);
        gpt2_write_to_checkpoint(model, filename_buffer);
    }
    snprintf(filename_buffer, sizeof(filename_buffer), "%s/state_%08d_%05d.bin", output_log_dir, step, rank);
    save_state(filename_buffer, step, model, train_loader);
    multi_gpu_barrier(multi_gpu_config);
    if (rank == 0) {
        snprintf(filename_buffer, sizeof(filename_buffer), "%s/DONE_%08d", output_log_dir, step);
        FILE* done_file = fopenCheck(filename_buffer, "w");
        fcloseCheck(done_file);
    }
}

void delete_checkpoint(const char* output_log_dir, int step, MultiGpuConfig* multi_gpu_config) {
    printf0("Deleting checkpoint at step %d from %s\n", step, output_log_dir);
    int rank = multi_gpu_config->process_rank;
    if (rank == 0) {
        snprintf(filename_buffer, sizeof(filename_buffer), "%s/model_%08d.bin", output_log_dir, step);
        remove(filename_buffer);
    }
    snprintf(filename_buffer, sizeof(filename_buffer), "%s/state_%08d_%05d.bin", output_log_dir, step, rank);
    remove(filename_buffer);
    if (rank == 0) {
        snprintf(filename_buffer, sizeof(filename_buffer), "%s/DONE_%08d", output_log_dir, step);
        remove(filename_buffer);
    }
}

#ifndef TESTING
// if we are TESTING (see test_gpt2.cu), we'll skip everything below this point
void error_usage() {
    fprintf(stderr, "Usage:   ./train_gpt2cu [options]\n");
    fprintf(stderr, "Options:\n");
    fprintf(stderr, "  -i <string> train data filename pattern (default = dev/data/tinyshakespeare/tiny_shakespeare_train.bin)\n");
    fprintf(stderr, "  -j <string> val data filename pattern (default = dev/data/tinyshakespeare/tiny_shakespeare_val.bin)\n");
    fprintf(stderr, "  -e <string> input .bin filename or descriptor, see code comments as docs. (default = gpt2_124M_bf16.bin)\n");
    fprintf(stderr, "  -o <string> output log dir (default = NULL, no logging)\n");
    fprintf(stderr, "  -n <int>    write optimization checkpoints every how many steps? (default 0, don't)\n");
    fprintf(stderr, "  -y <int>    resume optimization found inside checkpoint dir? (0=restart/overwrite, 1=resume/append)\n");
    // ... (El resto de la ayuda de CLI permanece sin cambios) ...
    exit(EXIT_FAILURE);
}

int main(int argc, char *argv[]) {
    MPI_Init(&argc, &argv);

    const char* train_data_pattern = "dev/data/tinyshakespeare/tiny_shakespeare_train.bin";
    const char* val_data_pattern = "dev/data/tinyshakespeare/tiny_shakespeare_val.bin";
    const char* load_filename = "gpt2_124M_bf16.bin";
    const char* lr_scheduler_type = "cosine";
    const char* output_log_dir = NULL;
    const char* checkpoint_dir = "paralel_checkpoint";

    int checkpoint_every = 0;
    int checkpoints_keep = 0;
    int major_checkpoint_every = 0;
    int resume = 0;
    int B = 4;
    int T = 1024;
    int total_batch_size = -1;
    float learning_rate = 3e-4f;
    int log_gpu_every = -1;
    int warmup_iterations = 0;
    float final_learning_rate_frac = 1.0f;
    float weight_decay = 0.0f;
    float skip_update_lossz = 0.0f;
    float skip_update_gradz = 0.0f;
    int val_loss_every = 20;
    int val_max_steps = 20;
    int sample_every = 20;
    int genT = 64;
    int overfit_single_batch = 0;
    int max_steps = -1;
    int override_enable_tf32 = 1;
    int use_master_weights = 1;
    int gelu_fusion = -1;
    int recompute = 1;
    int zero_stage = 0;
    int hellaswag_eval = 0;
    int num_processes = 1;
    int process_rank = 0;
    int gpus_per_node = 8;
    char nccl_init_method[256] = "mpi";
    char server_ip[256] = "";
    char fs_path[256] = "";

    for (int i = 1; i < argc; i+=2) {
        if (i + 1 >= argc) { error_usage(); } if (argv[i][0] != '-') { error_usage(); } if (!(strlen(argv[i]) == 2 || strlen(argv[i]) == 3)) { error_usage(); }
        if (argv[i][1] == 'i') { train_data_pattern = argv[i+1]; } else if (argv[i][1] == 'j') { val_data_pattern = argv[i+1]; }
        else if (argv[i][1] == 'e') { load_filename = argv[i+1]; } else if (argv[i][1] == 'o') { output_log_dir = argv[i+1]; }
        else if (argv[i][1] == 'n' && argv[i][2] == '\0') { checkpoint_every = atoi(argv[i+1]); } else if (argv[i][1] == 'y') { resume = atoi(argv[i+1]); }
        else if (argv[i][1] == 'b') { B = atoi(argv[i+1]); } else if (argv[i][1] == 't') { T = atoi(argv[i+1]); }
        else if (argv[i][1] == 'd') { total_batch_size = atoi(argv[i+1]); } else if (argv[i][1] == 'l' && argv[i][2] == '\0') { learning_rate = atof(argv[i+1]); }
        else if (argv[i][1] == 'l' && argv[i][2] == 'g') { log_gpu_every = atoi(argv[i+1]); } else if (argv[i][1] == 'u') { warmup_iterations = atoi(argv[i+1]); }
        else if (argv[i][1] == 'q') { final_learning_rate_frac = atof(argv[i+1]); } else if (argv[i][1] == 'c') { weight_decay = atof(argv[i+1]); }
        else if (argv[i][1] == 'x') { max_steps = atoi(argv[i+1]); } else if (argv[i][1] == 'v') { val_loss_every = atoi(argv[i+1]); }
        else if (argv[i][1] == 'm') { val_max_steps = atoi(argv[i+1]); } else if (argv[i][1] == 's' && argv[i][2] == '\0') { sample_every = atoi(argv[i+1]); }
        else if (argv[i][1] == 'g' && argv[i][2] == 'e') { gelu_fusion = atoi(argv[i+1]); } else if (argv[i][1] == 'g') { genT = atoi(argv[i+1]); }
        else if (argv[i][1] == 'a') { overfit_single_batch = atoi(argv[i+1]); } else if (argv[i][1] == 'f') { override_enable_tf32 = atoi(argv[i+1]); }
        else if (argv[i][1] == 'w') { use_master_weights = atoi(argv[i+1]); } else if (argv[i][1] == 'z') { zero_stage = atoi(argv[i+1]); }
        else if (argv[i][1] == 'r') { recompute = atoi(argv[i+1]); } else if (argv[i][1] == 'h') { hellaswag_eval = atoi(argv[i+1]); }
        else if (argv[i][1] == 'k') { lr_scheduler_type = argv[i+1]; } else if (argv[i][1] == 'p' && argv[i][2] == 'i') { strcpy(nccl_init_method, argv[i+1]); }
        else if (argv[i][1] == 'p' && argv[i][2] == 'f') { strcpy(fs_path, argv[i+1]); } else if (argv[i][1] == 'p' && argv[i][2] == 's') { strcpy(server_ip, argv[i+1]); }
        else if (argv[i][1] == 'p' && argv[i][2] == 'n') { num_processes = atoi(argv[i+1]); } else if (argv[i][1] == 'p' && argv[i][2] == 'r') { process_rank = atoi(argv[i+1]); }
        else if (argv[i][1] == 'p' && argv[i][2] == 'g') { gpus_per_node = atoi(argv[i+1]); } else if (argv[i][1] == 's' && argv[i][2] == 'l') { skip_update_lossz = atof(argv[i+1]); }
        else if (argv[i][1] == 's' && argv[i][2] == 'g') { skip_update_gradz = atof(argv[i+1]); } else if (argv[i][1] == 'n' && argv[i][2] == 'k') { checkpoints_keep = atoi(argv[i+1]); }
        else if (argv[i][1] == 'n' && argv[i][2] == 'm') { major_checkpoint_every = atoi(argv[i+1]); } else { error_usage(); }
    }

    multi_gpu_config = multi_gpu_config_init(num_processes, process_rank, gpus_per_node, server_ip, fs_path, nccl_init_method);
    common_start(override_enable_tf32, false);

    assert(warmup_iterations >= 0);
    int tokens_per_fwdbwd = B * T * multi_gpu_config.num_processes;
    if (total_batch_size == -1) { total_batch_size = tokens_per_fwdbwd; }
    if (gelu_fusion == -1) { gelu_fusion = 0; }
    assert(total_batch_size % tokens_per_fwdbwd == 0);
    int grad_accum_steps = total_batch_size / tokens_per_fwdbwd;
    if (overfit_single_batch == 1) { train_data_pattern = val_data_pattern; }

    // --- MODIFICACION: Lógica de CSV ---
    FILE* csv_file = NULL;
    if (multi_gpu_config.process_rank == 0) {
        char csv_filename[256];
        snprintf(csv_filename, sizeof(csv_filename), "paralel_%dgpu_metrics.csv", multi_gpu_config.num_processes);
        csv_file = fopen(csv_filename, "a");
        if (csv_file == NULL) {
            printf("Error abriendo archivo CSV para escritura.\n");
            MPI_Abort(MPI_COMM_WORLD, 1);
        }
        fseek(csv_file, 0, SEEK_END);
        if (ftell(csv_file) == 0) {
            fprintf(csv_file, "step,loss,computation_time_ms,communication_time_ms,total_host_time_ms,gflops_per_sec,mfu_percentage\n");
        }
    }
    // --- FIN MODIFICACION ---

    int resuming = 0;
    int resume_max_step = -1;
    if (checkpoint_every > 0) {
        resume_max_step = find_max_step(checkpoint_dir);
    }
    
    if (resume == 1) {
        if (resume_max_step != -1) {
            resuming = 1;
            snprintf(filename_buffer, sizeof(filename_buffer), "%s/model_%08d.bin", checkpoint_dir, resume_max_step);
        }
    }

    GPT2 model;
    gpt2_init_common(&model);
    if (resuming == 1) {
        bool weight_init = !use_master_weights;
        gpt2_build_from_checkpoint(&model, filename_buffer, weight_init);
    } else if (ends_with_bin(load_filename)) {
        gpt2_build_from_checkpoint(&model, load_filename);
    } else {
        gpt_build_from_descriptor(&model, load_filename);
    }

    model.use_master_weights = use_master_weights; model.gelu_fusion = gelu_fusion; model.recompute = recompute;
    DataLoader train_loader, val_loader;
    dataloader_init(&train_loader, train_data_pattern, B, T, multi_gpu_config.process_rank, multi_gpu_config.num_processes, (overfit_single_batch == 1) ? 0 : 1);
    dataloader_init(&val_loader, val_data_pattern, B, T, multi_gpu_config.process_rank, multi_gpu_config.num_processes, 0);
    int train_num_batches = max_steps;
    if (train_num_batches == -1) { train_num_batches = train_loader.num_tokens / total_batch_size; }
    int val_num_batches = val_max_steps;
    if (val_num_batches == -1) { val_num_batches = val_loader.num_tokens / tokens_per_fwdbwd; }
    EvalLoader eval_loader;
    const char* hellaswag_path = "dev/data/hellaswag/hellaswag_val.bin";
    const bool hellaswag_available = access(hellaswag_path, F_OK) == 0;
    const bool run_hellaswag = hellaswag_eval && hellaswag_available;
    if (run_hellaswag) { evalloader_init(&eval_loader, hellaswag_path, B, T, multi_gpu_config.process_rank, multi_gpu_config.num_processes); }
    set_zero_configs(&multi_gpu_config, zero_stage, model.num_parameters);
    if (multi_gpu_config.process_rank == 0 && output_log_dir != NULL) { create_dir_if_not_exists(output_log_dir); }
    Logger logger; logger_init(&logger, output_log_dir, multi_gpu_config.process_rank, resume);
    Tokenizer tokenizer; tokenizer_init(&tokenizer, "gpt2_tokenizer.bin");
    LearningRateScheduler lr_scheduler; lr_scheduler_init(&lr_scheduler, lr_scheduler_type, learning_rate, warmup_iterations, train_num_batches, final_learning_rate_frac);
    int* gen_tokens = (int*)mallocCheck(B * T * sizeof(int));
    floatX* cpu_logits_raw = (floatX*)mallocCheck(model.config.vocab_size * sizeof(floatX));
    float* cpu_logits = (float*)mallocCheck(model.config.vocab_size * sizeof(float));

    int step = 0;
    gpt2_allocate_state(&model, B, T);
    if (resuming == 1) {
        snprintf(filename_buffer, sizeof(filename_buffer), "%s/state_%08d_%05d.bin", checkpoint_dir, resume_max_step, multi_gpu_config.process_rank);
        load_state(&step, &model, &train_loader, filename_buffer);
    }

    OutlierDetector loss_outlier_detector, grad_norm_outlier_detector;
    init_detector(&loss_outlier_detector); init_detector(&grad_norm_outlier_detector);
    assert(T <= model.config.max_seq_len);
    
    hipEvent_t start, end;
    cudaCheck(hipEventCreate(&start));
    cudaCheck(hipEventCreate(&end));
    double mpi_start_time, mpi_end_time;
    cudaCheck(hipProfilerStart());
    double total_sum_iteration_time_s = 0.0;
    float ema_tokens_per_second = 0.0f;

    for (; step <= train_num_batches; step++) {
        // Validation and text generation logic here...
        int last_step = step == train_num_batches;
        if (step % val_loss_every == 0 || last_step) { /* ... */ }
        if (run_hellaswag && ((step > 0 && step % val_loss_every == 0) || last_step)) { /* ... */ }
        if (multi_gpu_config.process_rank == 0 && sample_every > 0 && (step > 0 && (step % sample_every) == 0 || last_step)) { /* ... */ }
        
        if (checkpoint_every > 0 && ((step > 0 && step % checkpoint_every == 0) || last_step)) {
            if (multi_gpu_config.process_rank == 0) { create_dir_if_not_exists(checkpoint_dir); }
            multi_gpu_barrier(&multi_gpu_config);
            write_checkpoint(checkpoint_dir, step, &model, &train_loader, &multi_gpu_config);
            int step_delete = step - checkpoints_keep * checkpoint_every;
            if (checkpoints_keep > 0 && step_delete > 0 && (major_checkpoint_every == 0 || step_delete % major_checkpoint_every != 0)) {
                delete_checkpoint(checkpoint_dir, step_delete, &multi_gpu_config);
            }
        }
        resuming = 0;
        if (last_step) { break; }
        if (overfit_single_batch == 1) { dataloader_reset(&train_loader); }
        
        multi_gpu_barrier(&multi_gpu_config);
        mpi_start_time = MPI_Wtime();
        cudaCheck(hipEventRecord(start));
        for (int micro_step = 0; micro_step < grad_accum_steps; micro_step++) {
            dataloader_next_batch(&train_loader);
            gpt2_forward(&model, train_loader.inputs, B, T);
            gpt2_backward_and_reduce(&model, train_loader.inputs, train_loader.targets, grad_accum_steps, micro_step);
        }
        float local_loss = model.mean_loss;
        float global_loss = 0.0f;
        MPI_Allreduce(&local_loss, &global_loss, 1, MPI_FLOAT, MPI_SUM, MPI_COMM_WORLD);
        global_loss /= multi_gpu_config.num_processes;
        model.mean_loss = global_loss;
        float zloss = (float)(update_detector(&loss_outlier_detector, (double)model.mean_loss));
        float step_learning_rate = get_learning_rate(&lr_scheduler, step);
        float grad_norm = gpt2_calculate_grad_norm(&model, &multi_gpu_config);
        float zgrad = (float)(update_detector(&grad_norm_outlier_detector, (double)grad_norm));
        if (isfinite(zloss) && skip_update_lossz != 0.0f && zloss > skip_update_lossz) { printf0("skipping update due to loss z-score of %f\n", zloss); }
        else if (isfinite(zgrad) && skip_update_gradz != 0.0f && zgrad > skip_update_gradz) { printf0("skipping update due to grad z-score of %f\n", zgrad); }
        else {
            float grad_clip = 1.0f;
            float grad_scale = (grad_norm > grad_clip) ? grad_clip / grad_norm : 1.0f;
            gpt2_update(&model, step_learning_rate, 0.9f, 0.95f, 1e-8f, weight_decay, grad_scale, step+1, &multi_gpu_config);
        }
        cudaCheck(hipEventRecord(end));
        cudaCheck(hipEventSynchronize(end));
        mpi_end_time = MPI_Wtime();

        float computation_time_ms;
        cudaCheck(hipEventElapsedTime(&computation_time_ms, start, end));
        double total_host_time_ms = (mpi_end_time - mpi_start_time) * 1000.0;
        double communication_time_ms = total_host_time_ms > computation_time_ms ? total_host_time_ms - computation_time_ms : 0.0;
        size_t tokens_processed = (size_t)multi_gpu_config.num_processes * B * T * grad_accum_steps;
        float tokens_per_second = tokens_processed / (computation_time_ms / 1000.0f);
        float bias_corrected_ema_tokens_per_second = tokens_per_second;
        if (step > 0) {
            total_sum_iteration_time_s += computation_time_ms / 1000.0f;
            ema_tokens_per_second = 0.95f * ema_tokens_per_second + 0.05f * tokens_per_second;
            bias_corrected_ema_tokens_per_second = ema_tokens_per_second / (1.0f - powf(0.95f, step));
        }
        float mfu = gpt2_estimate_mfu(&model, B * T * grad_accum_steps, computation_time_ms / 1000.0f);
        long long flops_per_token = 6LL * model.num_parameters + (size_t)6 * model.config.num_layers * model.config.channels * T;
        long long total_flops_per_step = flops_per_token * B * T * grad_accum_steps;
        double gflops_per_sec = (computation_time_ms > 0) ? (total_flops_per_step / (computation_time_ms / 1000.0)) / 1e9 : 0.0;

        printf0("step %4d/%d | loss %7.6f | norm %6.4f | lr %.2e | comp_t %.2fms | comm_t %.2fms | mfu %.1f%% | tps %.0f\n",
                step + 1, train_num_batches, model.mean_loss, grad_norm, step_learning_rate,
                computation_time_ms, communication_time_ms, 100*mfu, bias_corrected_ema_tokens_per_second);

        if (multi_gpu_config.process_rank == 0 && csv_file != NULL) {
            fprintf(csv_file, "%d,%f,%f,%f,%f,%f,%f\n", step + 1, model.mean_loss, computation_time_ms, communication_time_ms, total_host_time_ms, gflops_per_sec, mfu*100);
            fflush(csv_file);
        }

        if (log_gpu_every > 0 && (step + 1) % log_gpu_every == 0) { /* ... */ }
        if (step == 3) { hipProfilerStop(); }
    }
    
    printf0("total average iteration time: %f ms\n", total_sum_iteration_time_s / (train_num_batches-1) * 1000);

    if (multi_gpu_config.process_rank == 0 && csv_file != NULL) { fclose(csv_file); }
    cudaCheck(hipEventDestroy(end));
    cudaCheck(hipEventDestroy(start));
    if (run_hellaswag) { evalloader_free(&eval_loader); }
    dataloader_free(&train_loader);
    dataloader_free(&val_loader);
    tokenizer_free(&tokenizer);
    free(cpu_logits_raw);
    free(cpu_logits);
    free(gen_tokens);
    multi_gpu_config_free(&multi_gpu_config);
    gpt2_free(&model);
    common_free(model);
    
    MPI_Finalize();
    return 0;
}
#endif
